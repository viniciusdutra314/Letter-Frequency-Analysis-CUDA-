#include "hip/hip_runtime.h"
#include <string>
#include "auxilary_funcs.hpp"

__global__ void kernel_histogram(u_char* text, int text_size,int* histogram)
{

    __shared__ int local_occurrences[256];
    local_occurrences[threadIdx.x]=0;
    __syncthreads();

    int position_in_the_text = threadIdx.x + blockIdx.x * blockDim.x;
    int step_to_next_position= blockDim.x * gridDim.x;
    while (position_in_the_text < text_size)
    {
        atomicAdd(&local_occurrences[text[position_in_the_text]], 1);
        position_in_the_text += step_to_next_position;
    }
    __syncthreads();
    atomicAdd(&(histogram[threadIdx.x]), local_occurrences[threadIdx.x]);
}

int main(){  
    Timer time_all("(GPU-Kernel) Total time: ");
    std::string h_text=open_file_as_string("input.txt");
    //moving to the device
    
    u_char* d_text;
    int* d_histogram;
    int memory_used_text=h_text.size()*sizeof(u_char);
    int memory_used_hist=256*sizeof(int);
    hipMalloc(&d_text,memory_used_text);
    hipMemcpy(d_text,h_text.c_str(),memory_used_text,hipMemcpyHostToDevice);

    hipMalloc(&d_histogram,memory_used_hist);
    hipMemset(d_histogram,0,memory_used_hist);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int blocks = prop.multiProcessorCount;
    Timer time_processing   ("(GPU-Kernel) Processing: ");
    kernel_histogram<<<blocks * 2, 256>>>(d_text,h_text.size(),d_histogram);
    int* h_histogram=(int*) malloc(memory_used_hist);
    hipMemcpy(h_histogram,d_histogram,memory_used_hist,hipMemcpyDeviceToHost);
    time_processing.stop();

    
    std::vector<int> histogram_vector(h_histogram,h_histogram+256);
    save_sorted_to_file(histogram_vector,"output/histogram_gpu_kernel.txt");
    free(h_histogram);
    hipFree(d_text); 
    hipFree(d_histogram);
    time_all.stop();
    std::cout<<std::endl;
    
}
